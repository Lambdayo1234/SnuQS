#include "api.h"

#include <hip/hip_runtime.h>

namespace snurt {

int GetDeviceCount() {
  hipError_t err;
  int count;
  err = hipGetDeviceCount(&count);
  if (err != hipSuccess)
    return -EINVAL;
  return count;
}

addr_t MallocHost(size_t count) {
  addr_t addr;
  hipError_t err;
  void *ptr;
  err = hipHostMalloc(&ptr, count);
  addr.ptr = (err == hipSuccess) ? ptr : nullptr;
  return addr;
}

addr_t MallocDevice(size_t count, size_t devno) {
  addr_t addr;
  hipError_t err;
  err = hipSetDevice(devno);
  if (err != hipSuccess) {
    addr.ptr = nullptr;
    return addr;
  }

  void *ptr;
  err = hipMalloc(&ptr, count);
  addr.ptr = (err == hipSuccess) ? ptr : nullptr;
  return addr;
}

} // namespace snurt
